#include "hip/hip_runtime.h"
#include "Solver.h"
#include "SolveImpl.h"
#include "Utils.cuh"

#include <hip/hip_runtime.h>
#include <iostream>

namespace uni
{
	void initial_device()
	{
		int count = 0;
		hipDeviceProp_t prop;
		hipGetDeviceCount(&count);
		std::cout << "Device count " << count << std::endl;
		for (int i = 0; i < count; ++i)
		{
			hipGetDeviceProperties(&prop, i);
			std::cout << "max threads per block " << prop.maxThreadsPerBlock << std::endl;
			std::cout << "max threads dim " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << std::endl;
			std::cout << "max grid size " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << std::endl;
		}

		checkCudaErrors(hipSetDevice(0));
	}

	void alloc_cuda_memory(SolverData * data, unsigned int p_size, unsigned int constraint_size)
	{
		checkCudaErrors(hipMalloc((void**)&data->p, p_size * sizeof(float3)));
		checkCudaErrors(hipMalloc((void**)&data->x, p_size * sizeof(float3)));
		checkCudaErrors(hipMalloc((void**)&data->v, p_size * sizeof(float3)));
		checkCudaErrors(hipMalloc((void**)&data->inv_m, p_size * sizeof(float)));
		checkCudaErrors(hipMalloc((void**)&data->cons, constraint_size * sizeof(DistCons)));
	}

	void set_positions(SolverData * data, float3 * host_positions, unsigned int p_size)
	{
		checkCudaErrors(hipMemcpy(data->p, host_positions, p_size * sizeof(float3), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(data->x, host_positions, p_size * sizeof(float3), hipMemcpyHostToDevice));
	}

	void set_velocities(SolverData * data, float3 * host_velocities, unsigned int p_size)
	{
		checkCudaErrors(hipMemcpy(data->v, host_velocities, p_size * sizeof(float3), hipMemcpyHostToDevice));
	}

	void set_inv_masses(SolverData * data, float * host_inv_m, unsigned int p_size)
	{
		checkCudaErrors(hipMemcpy(data->inv_m, host_inv_m, p_size * sizeof(float), hipMemcpyHostToDevice));
	}

	void set_constraints(SolverData * data, DistCons * host_constraints, unsigned int constraint_size)
	{
		checkCudaErrors(hipMemcpy(data->cons, host_constraints, constraint_size * sizeof(DistCons), hipMemcpyHostToDevice));
	}

	void solve(SolverData * data, unsigned int p_size, unsigned int cons_size, float time_step, int iter_cnt)
	{
		solve_Gauss(data, p_size, cons_size, time_step, iter_cnt);
	}

	void get_positions(SolverData * data, float3 * host_positions, unsigned int p_size)
	{
		checkCudaErrors(hipMemcpy(host_positions, data->x, p_size * sizeof(float3), hipMemcpyDeviceToHost));
	}


	void free_cuda_memory(SolverData * data)
	{
		if (data == nullptr) return;

		checkCudaErrors(hipFree(data->p));
		checkCudaErrors(hipFree(data->x));
		checkCudaErrors(hipFree(data->v));
		checkCudaErrors(hipFree(data->inv_m));
		checkCudaErrors(hipFree(data->cons));
	}


	void reset_device()
	{
		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		checkCudaErrors(hipDeviceReset());
	}

}
