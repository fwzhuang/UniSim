#include "hip/hip_runtime.h"
#include "SolveImpl.h"
#include "Solver.h"
#include "Utils.cuh"
#include "GraphColoring.cuh"
#include "Collision.h"
#include "hip/hip_runtime_api.h"

#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>
#include <hip/device_functions.h>

namespace uni
{
	__device__ float eps = 1e-20f;

	int threadsPerBlock = 512;

	template <typename AdjItemType>
	__global__ void adjTable_k(DistanceConstraint * cons, AdjItemType * adj_cons, int con_size)
	{
		unsigned int cid = threadIdx.x + blockDim.x * blockIdx.x;
		if (cid >= con_size) return;

		//is_colored[cid] = 0;

		//palettes[cid].valid_size = palettes[cid].max_size * 0.94f;
		//for (int i = 0; i < palettes[cid].valid_size; ++i)
		//	palettes[cid].colors[i] = 1;

		int pid0 = cons[cid].pid.x;
		int pid1 = cons[cid].pid.y;

		adj_cons[cid].valid_size = 0;
		for (int i = 0; i < con_size; ++i)
		{
			if (cid != i &&
				(cons[i].pid.x == pid0 || cons[i].pid.x == pid1 || cons[i].pid.y == pid0 || cons[i].pid.y == pid1))
			{
				adj_cons[cid].adjs[adj_cons[cid].valid_size] = i;
				adj_cons[cid].valid_size += 1;
			}
		}
	}

	__global__ void freeRun_Gauss_k(float3 * x, float3 * p, float3 * v, float * inv_m, float time_step, unsigned int p_size)
	{
		unsigned int pid = threadIdx.x + blockDim.x * blockIdx.x;
		if (pid >= p_size) return;

		float3 force = { 0.0f, -0.010f, 0.0f };

		float3 offset = 0.5f * inv_m[pid] * time_step * force;
		v[pid] = v[pid] + offset;
		p[pid] = x[pid] + v[pid] * time_step;
	}

	__global__ void projectConstraint_Gauss_k(float3 * x, float3 * p, float3 * v, float * inv_m, DistanceConstraint * cons, int * colors, unsigned int con_size, int gid)
	{
		unsigned int cid = threadIdx.x + blockDim.x * blockIdx.x;
		if (cid >= con_size) return;
		if (colors[cid] != gid) return;

		int pid0 = cons[cid].pid.x;
		int pid1 = cons[cid].pid.y;
		float d = cons[cid].d;

		float inv_m0 = inv_m[pid0];
		float inv_m1 = inv_m[pid1];

		if (inv_m0 + inv_m1 < eps) return;

		float3 p0 = p[pid0];
		float3 p1 = p[pid1];

		float dist = length(p0 + (-p1));
		if (-eps < dist && dist < eps) return;
		float delta_d = dist - d;

		//float3 tmp = (delta_d / (inv_m0 + inv_m1)) * (p0 + (-p1));
		float3 tmp = delta_d / ((inv_m0 + inv_m1) * dist) * (p0 + (-p1));
		//float3 tmp = (1.0f / (inv_m0 + inv_m1) * delta_d) * (p0 + (-p1));
		float3 d_p0 = -inv_m0 * tmp;
		float3 d_p1 = inv_m1 * tmp;

		p[pid0] = p[pid0] + d_p0;
		p[pid1] = p[pid1] + d_p1;
	}

	__global__ void updateState_Gauss_k(float3 * x, float3 * p, float3 * v, float * inv_m, float time_step, unsigned int p_size)
	{
		unsigned int pid = threadIdx.x + blockDim.x * blockIdx.x;
		if (pid >= p_size) return;

		v[pid] = (p[pid] + (-x[pid])) * (1.0f / time_step);
		x[pid] = p[pid];
	}

	template <int MaxDegree>
	void callGraphColoring_Gauss(SolverData * data, int * cons_colors, unsigned int cons_size)
	{
		static AdjItem<MaxDegree> * adj_table = nullptr;
		if (adj_table == nullptr)
			hipMalloc((void **)&adj_table, cons_size * sizeof(AdjItem<MaxDegree>));

		int threadsPerBlock = 1024;

		dim3 con_blocks((cons_size + threadsPerBlock - 1) / threadsPerBlock);
		dim3 con_threads(threadsPerBlock);

		adjTable_k<AdjItem<MaxDegree>> << <con_blocks, con_threads >> > (data->cons, adj_table, cons_size);
		getLastCudaError("Kernel execution failed");
		checkCudaErrors(hipDeviceSynchronize());

		graph_coloring<MaxDegree>(adj_table, cons_colors, cons_size);
	}

	void solve_Gauss(SolverData * data, unsigned int p_size, unsigned int cons_size, float time_step, int iter_cnt)
	{
		static int * colors = nullptr;

#ifdef PROFILE_CUDA
		hipEvent_t start, stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));
		checkCudaErrors(hipEventRecord(start, 0));
#endif

		int threadsPerBlock = 1024;

		dim3 p_blocks((p_size + threadsPerBlock - 1) / threadsPerBlock);
		dim3 p_threads(threadsPerBlock);
		
		dim3 con_blocks((cons_size + threadsPerBlock - 1) / threadsPerBlock);
		dim3 con_threads(threadsPerBlock);

		freeRun_Gauss_k <<<p_blocks, p_threads>>> (data->x, data->p, data->v, data->inv_m, time_step, p_size);
		getLastCudaError("Kernel execution failed");
		checkCudaErrors(hipDeviceSynchronize());

		if (colors == nullptr)
		{
			hipMalloc((void **)&colors, cons_size * sizeof(int));
			callGraphColoring_Gauss<16>(data, colors, cons_size);
		}
		for (int i = 0; i < iter_cnt; ++i)
		{
			for (int gid = 0; gid < 16; ++gid)
			{
				projectConstraint_Gauss_k << <con_blocks, con_threads >> >(data->x, data->p, data->v, data->inv_m, data->cons, colors, cons_size, gid);
				getLastCudaError("Kernel execution failed");
				checkCudaErrors(hipDeviceSynchronize());
			}
		}

		CollideGridSpace collide_space{ { -50.0f, -50.0f, -50.0f },{ 50.0f, 50.0f, 50.0f }, 0.5f };
		solveCollision(collide_space, data->p, data->inv_m, p_size, 0.4f, 2.0f * iter_cnt);

		updateState_Gauss_k <<<p_blocks, p_threads>>>(data->x, data->p, data->v, data->inv_m, time_step, p_size);
		getLastCudaError("Kernel execution failed");
		checkCudaErrors(hipDeviceSynchronize());

#ifdef PROFILE_CUDA
		checkCudaErrors(hipEventRecord(stop, 0));
		
		checkCudaErrors(hipEventSynchronize(stop));
		float elapse_time;
		checkCudaErrors(hipEventElapsedTime(&elapse_time, start, stop));
		std::cout << "solve on GPU time " << elapse_time << " ms" << std::endl;
		
		checkCudaErrors(hipEventDestroy(start));
		checkCudaErrors(hipEventDestroy(stop));
#endif
	}

}

